#include "synchronoussgd.h"

void crossbowKernelOptimiserSynchronousSGD (crossbowStreamP s) {
	
	float minusone = -1;
	
	/* Number of model (and gradient) parameters */
	int elements = s->model->elements;

	/* Get replica model data buffer */
	crossbowDataBufferP model = s->model->data;

	/* Get replica's gradient data buffer */
	crossbowDataBufferP gradient = s->model->gradient;

	/* Get base model's gradient data buffer */
	crossbowDataBufferP theGradient = s->theModel->gradient;
	
	/* Apply weight decay to gradient, if set */
	if (s->model->conf->weightDecay > 0) {
		checkCublasStatus(hipblasSaxpy (s->cublasHandle[s->op->branch], 
			elements, 
			&(s->model->conf->weightDecay), 
			(float *) (model->dev), 1, 
			(float *) (gradient->dev), 1));
	}
	
	/* For debugging purposes
	 * 
	 * checkCudaErrors(hipDeviceSynchronize());
	 * 
	 * float checksum = crossbowDataBufferComputeCheckSum(gradient, 0, s->model->bytes);
	 * info("Gradient checksum of task %d is %.5f\n", s->task, checksum);
	 */
	
	/* Record event that gradient is ready to be used */
	checkCudaErrors(hipEventRecord (s->model->client, s->stream[s->op->branch]));
	
	/* Accumulate replica's gradient to base model's gradient, applying learning rate */
	checkCudaErrors(hipStreamWaitEvent(s->modelSynchronisationStream, s->model->client, 0));
	
	if (s->model->conf->momentumMethod == NESTEROV) {
		err("Nesterov's momentum has been disabled\n");
	}
	
	float rate = minusone * crossbowSolverConfGetLearningRate (s->model->conf, s->task);
	checkCublasStatus(hipblasSaxpy (s->modelSynchronisationHandle, 
		elements, 
		&(rate), 
		(float *) (gradient->dev), 1, 
		(float *) (theGradient->dev), 1));

	/* Record event that replica model gradient is no longer required */
	checkCudaErrors(hipEventRecord(s->model->server, s->modelSynchronisationStream));

	return;
}
